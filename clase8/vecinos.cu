#include "hip/hip_runtime.h"
// main.cu 
#include <iostream> 
#include <ctime> 
#include <hip/hip_runtime.h>
#include <>

#include <hip/hip_runtime.h> // For float3 structure

using namespace std; 

__global__ void FindClosetsGPU(float3* points, int* indices, int count){
	
	if(count<=1) return;

	int idx= threadIdx.x + blockDim.x * blockIdx.x;

	if(idx < count){

		float distToClosets= 3.40282e38f;

		for (int i=0;i<count;i++){

			if(i==idx) continue;

			float dist= ((points.x[idx]-points[i].x)*(points.x[idx]-points[i].x) + (points.y[idx]-points[i].y)*(points.y[idx]-points[i].y) + (points.z[idx]-points[i].z)*(points.z[idx]-points[i].z));

			if(dist<distToClosets){
				distToClosets=dist;
				indices[idx]=i;
			}

		}



	}
}



int main(){
	
	const int count=10000;

	int *indexOfClosest= new int[count];

	float3 *points=new float3[count];
	float3 *d_points;
	int *d_indexOfClosest;

	for(int i=0;i<count;i++){

		points[i].x=(float)((rand()%1000)-5000);
		points[i].y=(float)((rand()%1000)-5000);
		points[i].z=(float)((rand()%1000)-5000);
	}


	hipMalloc(&d_points,sizeof(float3)*count);
	hipMemcpy(d_points,points,sizeof(float3)*count,hipMemcpyHostToDevice);

	hipMalloc(&d_indexOfClosest,sizeof(int)*count);

	return 0;
}