#include <stdlib.h>
#include <stdio.h>
#include <time.h>
//#include "initial.h"
#include <hip/hip_runtime.h>

#define N 1024

__global__ void suma_GPU(float *a, float *b, float *c){

	int myID= threadIdx.x + blockDim.x * blockIdx.x;

	if(myID<N){
		c[myID]= a[myID]+ b[myID];
	}
}

void OneD_InitialData(float *ip, int size){

	for(int i=0; i<size;i++){

		ip[i]= (float) rand()/10.0;
	}

}

void print1D_arrays(float *pi,int size){

	for(int i=0;i<size;i++){
		printf("%f\n", pi[i]);
	}
}

int main(){

	size_t nBytes= N* sizeof(float);

	float *h_a, *h_b, *h_c;
	float *d_a, *d_b, *d_c;

	h_a=(float *)malloc(nBytes);
	h_b=(float *)malloc(nBytes);
	h_c=(float *)malloc(nBytes);

	hipMalloc( (void**)&d_a,nBytes);
	hipMalloc( (void**)&d_b,nBytes);
	hipMalloc( (void**)&d_c,nBytes);


	OneD_InitialData(h_a,N);
	OneD_InitialData(h_b,N);

	hipMemcpy(d_a,h_a,nBytes,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,nBytes,hipMemcpyHostToDevice);

	int Blocks=32;

	int Nhilos= N/Blocks;

	suma_GPU<<<Blocks,Nhilos >>>(d_a,d_b,d_c);

	hipMemcpy(h_c,d_c,nBytes,hipMemcpyDeviceToHost);

	print1D_arrays(h_c,N);


	free(h_a);
	free(h_b);
	free(h_c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);







	return 0;
}