#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<stdio.h>
#include<time.h>
#include <hip/hip_runtime.h>

#define N 1024

__global__ void sumaGPU(float *a, float* b, float *c){
	
	int myID= threadIdx.x + blockDim.x * blockIdx.x;

	if(myID<N){
		c[myID]= a[myID]+ b[myID];
	}

}

void 1DInitialData(float *ip,int size){
	
	for(int i=0; i<size; i++){

		ip[i]= (float) rand()/10.0;
	}

}

void print1DData(float *ip,int size){
	
	for(int i=0; i<size; i++){

		printf(" %f\n", ip[i]);
	}

}



int main(){
	
	size_t nBytes= N* sizeof(float);

	float *h_a, *h_b, *h_c;
	float *d_a, *d_b, *d_c;

 	// asignamos memoria en el host

 	h_a= (float *)malloc(nBytes);
	h_b= (float *)malloc(nBytes);
	h_c= (float *)malloc(nBytes);

	//asignamos memoria en el device

	hipMalloc( (void**)&d_a,nBytes);
	hipMalloc( (void**)&d_b,nBytes);
	hipMalloc( (void**)&d_c,nBytes);

	// inicializar los arreglos

	1DInitialData(h_a,N);
	1DInitialData(h_b,N);

	// mando los datos a la ldevice

	hipMemcpy(d_a,h_a,nBytes,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,nBytes,hipMemcpyHostToDevice);

	// ------------ procesamiento ----------------- //

	int Blocks= 32;

	int Nhilos= N/ Blocks; 

	sumaGPU<<< Blocks, Nhilos>>>(d_a,d_b,d_c);


	// ------------ procesamiento ----------------- //

	hipMemcpy(h_c,d_c,nBytes,hipMemcpyDeviceToHost);

	print1DData(h_c,N);

	free(h_a);
	free(h_b);
	free(h_c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0; 
}