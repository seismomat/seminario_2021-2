#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include<fstream>

#define Pi 3.141516
#define Nthreads 32 

using namespace std; 

__global__ void Sinodails(double* cosine, double* sine, int tam){
	
		int Id= threadIdx.x + blockDim.x* blockIdx.x;

		if(Id<tam){

			for(int i=0;i<tam;i++){
				cosine[Id*tam+i]=cos((2 * Id * i * Pi) / tam);
				sine[Id*tam+i]= sin((2 * Id * i * Pi) / tam);
			}

		}
}

__device__ double GetAmp(double real, double im)
{
    return sqrt(real*real+im*im);
}

__global__ void DFT(double* signal, double* cosine, double* sine, int tam, double* spectrum){
	
	int Id= threadIdx.x + blockDim.x* blockIdx.x;

	extern __shared__ double Shasignal[];

	double temp1, temp2;

	Shasignal[threadIdx.x]=signal[threadIdx.x];

	 __syncthreads();


	 for(int i=0;i<tam;i++){
	 	temp1+= cosine[Id*tam +i]*Shasignal[i];
	 	
	 	temp2+= sine[Id*tam +i]*Shasignal[i];
	 }


	spectrum[Id]=GetAmp(temp1,temp2);
	__syncthreads();

	printf(" jaja");
}

void Signals(double *signal,double *time,int tam){
	
	double dt=0.02;

for(int i=0;i<tam;i++){
	double R1 = (double) rand() / (double) RAND_MAX;
	double R2 = (double) rand() / (double) RAND_MAX;

	signal[i] = (double) sqrt( -2.0f * log( R1 )) * cos( 2.0f * Pi * R2 );
	time[i]=i*dt;

	}


}

int main(){

	// host variables
	double *signal, *time;
	double *cosine, *sine;
	double *spectrum;
	int tam=256;
	size_t dBytes=tam*sizeof(double);
	size_t ddBytes=tam*tam*sizeof(double);

	// device variables 
	double *d_cosine, *d_sine;
	double *d_signal;
	double *d_spectrum;

	// kernel variables 

	int Blocks= tam/Nthreads;


	// -------------------------  BODY ------------------ /////

	cosine= (double*)malloc(ddBytes);
	sine=(double*)malloc(ddBytes);
	spectrum=(double*)malloc(dBytes);
	signal=(double*)malloc(dBytes);
	time=(double*)malloc(dBytes);

	Signals(signal,time,tam);


	/// ---------------------------------------------------

	// we allocate cosine and sine arrays

	hipMalloc((void**)&d_cosine,ddBytes);
	hipMalloc((void**)&d_sine,ddBytes);


	Sinodails<<<Blocks,Nthreads>>>(d_cosine,d_sine,tam);

	hipDeviceSynchronize();
	/// ---------------------------------------------------
	
	hipMalloc((void**)&d_spectrum,dBytes);
	hipMalloc((void**)&d_signal,dBytes);
	hipMemcpy(d_signal,signal,dBytes,hipMemcpyHostToDevice);

	DFT<<<Blocks,Nthreads,dBytes >>>(d_signal,d_cosine,d_sine,tam,d_spectrum);

	hipDeviceSynchronize();

	hipMemcpy(spectrum,d_spectrum,dBytes,hipMemcpyDeviceToHost);
	// -------------------------  BODY ------------------ /////


	/// ---------------

	ofstream file1,file2;
  	file1.open("fourier.dat");
  	file2.open("signal.dat");
    
    for (int i=0; i<tam;i++){
    	file1<<time[i]<<" "<<spectrum[i]<<endl;
    	file2<<time[i]<<" "<<signal[i]<<endl;
    }

  	
  /// ---------------
	ofstream file3;

	file3.open("imprimir.gnu");

	file3<<"set terminal eps transparent size 6,4 lw 1.8 enhanced font \"Times,24\""<<endl;
	file3<<"set encoding iso_8859_1"<<endl;
	file3<<"set title 'fourier'"<<endl;
	file3<<"set output \"imprimir.eps\""<<endl;
	file3<<"set grid"<<endl;
	file3<<"set xrange[0:5]"<<endl;
	file3<<"set yrange[0:10]"<<endl;
	//file3<<"set datafile separator whiteespace"<<endl;
	file3<<"plot 'fourier.dat' w l"<<endl;
	file3.close();

	system("gnuplot imprimir.gnu");
	system(" evince imprimir.eps");

  /// ----------------

    /// ---------------
	ofstream file4;

	file4.open("imprimir1.gnu");

	file4<<"set terminal eps transparent size 6,4 lw 1.8 enhanced font \"Times,24\""<<endl;
	file4<<"set encoding iso_8859_1"<<endl;
	file4<<"set title 'Senal'"<<endl;
	file4<<"set output \"imprimir1.eps\""<<endl;
	file4<<"set grid"<<endl;
	file4<<"set xrange[0:5]"<<endl;
	file4<<"set yrange[-3:3]"<<endl;
	//file3<<"set datafile separator whiteespace"<<endl;
	file4<<"plot 'signal.dat' w l"<<endl;
	file4.close();

	system("gnuplot imprimir1.gnu");
	system(" evince imprimir1.eps");

  /// ----------------
	hipFree(d_cosine);
	hipFree(d_sine);
	hipFree(d_signal);
	free(cosine);
	free(sine);
	free(signal);

	return 0; 
}


